#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#define OBJECT_COUNT 1500 //Do NOT go beyond 1500
#define MAX_MAPPINGS 100000000
#define BLOCK_DIM 1024
// includes, system
// #include <GL/glew.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <ctime>


#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <bits/stdc++.h>


#define GLM_FORCE_CUDA
#include "glm/glm.hpp"


#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// vbo variables
GLuint vbo;
GLuint ibo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBOAndIBO(GLuint *vbo, GLuint *ibo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBOAndIBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

struct object{
    int n_vertices;
    float4 speed;
}objects[OBJECT_COUNT];

float4 *host_pos; //globally declared to allot more vertices than feasible in local scope

std::vector<glm::vec3> vertices;
std::vector<unsigned int> mappings;

bool loadOBJ(
    const char * path, 
    std::vector<glm::vec3> & out_vertices, 
    std::vector <unsigned int> & mappings
);

__device__ int getObjectId(int index, struct object* d_objects){
    int sum = 0;
    for (int i = 0; i < OBJECT_COUNT; ++i)
    {
        sum = sum + d_objects[i].n_vertices;
        if(index < sum)
        {
            // if(i==0)
            // printf("Sent %d\n", i);
            return i;
        }

    }
    // printf("getObjectId: Object Id not found. %d %d %d %d Sending -1\n",index, sum, d_objects[0].n_vertices, d_objects[1].n_vertices);
    return -1;
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, struct object* d_objects, float time)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int object_id = getObjectId(idx, d_objects);
    if(object_id == -1)
        return;
    float4 speed = d_objects[object_id].speed;
    pos[idx] = make_float4(pos[idx].x + speed.x*time, pos[idx].z + speed.z*time, 
        pos[idx].y+speed.y*time, 1.0f);
}



void launch_kernel(float4 *pos, struct object* objects, float time)
{
    // execute the kernel
    int threadsInBlock = ceil((float)vertices.size()/BLOCK_DIM);
    dim3 grid(threadsInBlock,1);
    dim3 block(BLOCK_DIM,1);
    simple_vbo_kernel<<< grid, block>>>(pos, objects, time);
}

bool checkHW(char *name, const char *gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

int findGraphicsGPU(char *name)
{
    int nGraphicsGPU = 0;
    int deviceCount = 0;
    bool bFoundGraphics = false;
    char firstGraphicsName[256], temp[256];

    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("> FAILED %s sample finished, exiting...\n", sSDKsample);
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("> There are no device(s) supporting CUDA\n");
        return false;
    }
    else
    {
        printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
        printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

        if (bGraphics)
        {
            if (!bFoundGraphics)
            {
                strcpy(firstGraphicsName, temp);
            }

            nGraphicsGPU++;
        }
    }

    if (nGraphicsGPU)
    {
        strcpy(name, firstGraphicsName);
    }
    else
    {
        strcpy(name, "this hardware");
    }

    return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	host_pos = (float4 *)malloc(MAX_MAPPINGS*sizeof(float4));
	if(host_pos == NULL){
		printf("Error: Unable to allocate mapping memory on host.\n");
		exit(-1);
	}
	srand((int)time(0));
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }


    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // command line mode only
    if (ref_file != NULL)
    {
        // This will pick the best possible CUDA capable device
        int devID = findCudaDevice(argc, (const char **)argv);

        printf("ref_file not found\n");
        // create VBO
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, vertices.size()*sizeof(glm::vec3)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        printf("ref_file found\n");
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
        {
            if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
            {
                return false;
            }
        }
        else
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }

        // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBOAndIBO(&vbo, &ibo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
        runCuda(&cuda_vbo_resource);

        // start rendering mainloop
        glutMainLoop();
    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    // printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    for (int i = 0; i<vertices.size(); ++i)
    {
        host_pos[i] = make_float4(vertices[i].x,vertices[i].y, vertices[i].z,1.0f);
    }
    checkCudaErrors(hipMemcpy(dptr, host_pos, vertices.size()*sizeof(float4), hipMemcpyHostToDevice));
    struct object* d_objects;
    checkCudaErrors(hipMalloc(&d_objects, sizeof(objects)));
    checkCudaErrors(hipMemcpy(d_objects, objects, sizeof(objects), hipMemcpyHostToDevice));
    launch_kernel(dptr, d_objects, g_fAnim);
    checkCudaErrors(hipFree(d_objects));
    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));

    // execute the kernel
    launch_kernel((float4 *)d_vbo_buffer, objects, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}


float4 getRandomSpeed(){
	float normalizer = 1000.0;
	return make_float4((rand()%100)/normalizer,(rand()%100)/normalizer,(rand()%100)/normalizer,1.0f);
}

void appendObject(std::vector<glm::vec3> &vertices, std::vector<unsigned int> &mappings,
 std::vector<glm::vec3> &temp_vertices, std::vector<unsigned int> &temp_mappings){
	int startIndex = vertices.size();
	vertices.insert(vertices.end(), temp_vertices.begin(), temp_vertices.end());
	for (int i = 0; i < temp_mappings.size(); ++i)
	{
		mappings.push_back(temp_mappings[i] + startIndex);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBOAndIBO(GLuint *vbo, GLuint *ibo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);
    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);
    // initialize buffer object
    vertices.clear();
    mappings.clear();
    std::vector<glm::vec3> temp_vertices_cube;
    std::vector<glm::vec3> temp_vertices_cone;
    std::vector<unsigned int> temp_mappings_cube;
    std::vector<unsigned int> temp_mappings_cone;
    if(OBJECT_COUNT  > 0){
    	bool res;
	    res = loadOBJ("cube.obj", temp_vertices_cube, temp_mappings_cube);
	    assert(res);
	    res = loadOBJ("cone.obj", temp_vertices_cone, temp_mappings_cone);
	    assert(res);
	}
    
	for (int i = 0; i < OBJECT_COUNT; ++i)
    {
    	if(mappings.size()>MAX_MAPPINGS){
	    	printf("Error! Mappings more than the threshold at object number %d. Exiting.\n", i);
	    	exit(-1);
	    }
        if(i%2 == 0){
	        objects[i].n_vertices = temp_vertices_cube.size();
	        objects[i].speed = getRandomSpeed();
	        appendObject(vertices, mappings, temp_vertices_cube, temp_mappings_cube);
		}
		else
		{
	        objects[i].n_vertices = temp_vertices_cone.size();
	        objects[i].speed = getRandomSpeed();
	        appendObject(vertices, mappings, temp_vertices_cone, temp_mappings_cone);
	    }
	    
    }
    printf("Size of vertices = %d\n", vertices.size());
    printf("Size of mappings = %d\n", mappings.size());
    //Code crashes at the following line for object count > 1500
    glBufferData(GL_ARRAY_BUFFER, vertices.size() * sizeof(float4), &vertices[0], GL_DYNAMIC_DRAW);
    // glBufferData(GL_ARRAY_BUFFER, vertices.size() * sizeof(glm::vec3), &vertices[0], GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    printf("Object allocated in OpenGL\n");
    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();

    assert(ibo);
    // create buffer object
    glGenBuffers(1, ibo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, *ibo);
    // initialize buffer object
    
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, mappings.size() * sizeof(unsigned int), &mappings[0], GL_STATIC_DRAW);

    // glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

    SDK_CHECK_ERROR_GL();

}


////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBOAndIBO(GLuint *vbo, GLuint *ibo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    // glBindBuffer(1, *ibo);
    // glDeleteBuffers(1, ibo);

    *vbo = 0;
    *ibo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ibo);

    glVertexPointer(4, GL_FLOAT, 0, 0);



    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_INDEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    // glDrawSomething(GL_TRIANGLES, 0, vertices.size());
    // printf("mappings size: %d\n",mappings.size());
    glDrawElements(GL_TRIANGLES, mappings.size(), GL_UNSIGNED_INT, (const GLvoid *)0);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_INDEX_ARRAY);


    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBOAndIBO(&vbo, &ibo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}


bool loadOBJ(
    const char * path, 
    std::vector<glm::vec3> & out_vertices, std::vector<unsigned int> &temp_mappings
){
    printf("Loading OBJ file %s...\n", path);



    FILE * file = fopen(path, "r");
    if( file == NULL ){
        printf("Impossible to open the file ! Are you in the right path ? See Tutorial 1 for details\n");
        getchar();
        return false;
    }

    while( 1 ){

        char lineHeader[128];
        // read the first word of the line
        int res = fscanf(file, "%s", lineHeader);
        if (res == EOF)
            break; // EOF = End Of File. Quit the loop.

        // else : parse lineHeader
        
        if ( strcmp( lineHeader, "v" ) == 0 ){
            glm::vec3 vertex;
            fscanf(file, "%f %f %f\n", &vertex.x, &vertex.y, &vertex.z );
            out_vertices.push_back(vertex);
        }
        else if ( strcmp( lineHeader, "f" ) == 0 ){
            unsigned int vertexIndex[3];
            int matches = fscanf(file, "%d %d %d\n", &vertexIndex[0], &vertexIndex[1], &vertexIndex[2]);
            if (matches != 3){
                printf("File can't be read by our simple parser :-( Try exporting with other options\n");
                return false;
            }
            temp_mappings.push_back(vertexIndex[0]-1);
            temp_mappings.push_back(vertexIndex[1]-1);
            temp_mappings.push_back(vertexIndex[2]-1);
        }else{
            // Probably a comment, eat up the rest of the line
            char stupidBuffer[1000];
            fgets(stupidBuffer, 1000, file);
        }

    }

    return true;
}

